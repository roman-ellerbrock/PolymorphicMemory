//
// Created by Roman Ellerbrock on 6/30/22.
//

#include "cuMemory.hpp"
#include <hip/hip_runtime.h>
//#include <blas.hh>

namespace polymorphic {
//	blas::set_device( device );

	using d = double;
	using cd = complex<double>;

	template class cuMemory<double>;
	template class cuMemory<complex<double>>;


	template void transferToDevice(cuMemory<d>&, const hostMemory<d>&);
	template void transferToDevice(cuMemory<cd>&, const hostMemory<cd>&);
	template void transferToHost(hostMemory<d>&, const cuMemory<d>&);
	template void transferToHost(hostMemory<cd>&, const cuMemory<cd>&);


}
