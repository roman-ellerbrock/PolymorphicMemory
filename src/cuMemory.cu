//
// Created by Roman Ellerbrock on 6/30/22.
//

#include "cuMemory.hpp"
#include <hip/hip_runtime.h>
//#include <blas.hh>

namespace polymorphic {
//	blas::set_device( device );

	using d = double;
	using cd = complex<double>;

	template class cudaAllocator<double>;
	template class cudaAllocator<complex<double>>;

	template class cuMemory<double>;
	template class cuMemory<complex<double>>;
}


